#include "lights.cu"
#pragma once

class Shape;

class AppendableArray {
public:
	void** values;

	int length;
	__host__
	void AddElement(void* ptr) {
		void** newVals;
		hipMallocManaged(&newVals, sizeof(Uint32) * (length + 1));
		for (int i = 0; i < length; i++) {
			newVals[i] = values[i];
		}
		newVals[length] = ptr;
		void** stor = values;
		values = newVals;
		hipFree(stor);
		length++;
	}
	__host__ __device__
	void* GetElement(int index) {
		return values[index];
	}
};

class ShapeHolder : public AppendableArray {
public:
	__host__ __device__
	Shape* GetShape(int index) {
		return (Shape*)values[index];
	}
};


class LightHolder : public AppendableArray {
public:
	__host__
	void AddLight(Vector3 pos, float intensity) {
		Light* ptr;
		hipMallocManaged(&ptr, sizeof(Light));
		ptr->pos = pos;
		ptr->intensity = intensity;
		AddElement(ptr);
	}
	__host__ __device__
	Light* GetLight(int index) {
		return (Light*)values[index];
	}
};
