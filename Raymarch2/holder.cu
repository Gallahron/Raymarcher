class AppendableArray {
public:
	void** values;

	int length;
	__host__
	void AddElement(void* ptr) {
		void** newVals;
		hipMallocManaged(&newVals, sizeof(Uint32) * (length + 1));
		for (int i = 0; i < length; i++) {
			newVals[i] = values[i];
		}
		newVals[length] = ptr;
		void** stor = values;
		values = newVals;
		hipFree(stor);
		length++;
	}
	__host__ __device__
	void* GetElement(int index) {
		return values[index];
	}
};

class ShapeHolder : public AppendableArray {
public:
	__host__
		Shape* CreateSphere(Vector3 pos, float rad, int blended) {
		Sphere* ptr;
		hipMallocManaged(&ptr, sizeof(Sphere));
		*ptr = Sphere(pos, 0.6f, blended);
		AddElement(ptr);
		return ptr;
	}
	__host__
		Shape* CreatePlane(float height, int blended) {
		Plane* ptr;
		hipMallocManaged(&ptr, sizeof(Plane));
		*ptr = Plane(height, blended);
		AddElement(ptr);
		return ptr;
	}
	__host__
		Shape* CreateCube(Vector3 pos, Vector3 rot, Vector3 bounds, int blended) {
		Cube* ptr;
		hipMallocManaged(&ptr, sizeof(Cube));
		*ptr = Cube(pos, rot, bounds, blended);

		AddElement(ptr);
		return ptr;
	}
	__host__
		Shape* CreateHollowCube(Vector3 pos, Vector3 rot, Vector3 bounds, float thickness, int blended) {
		HollowCube* ptr;
		hipMallocManaged(&ptr, sizeof(Cube));
		*ptr = HollowCube(pos, rot, bounds, thickness, blended);

		AddElement(ptr);
		return ptr;
	}
	__host__ __device__
	Shape* GetShape(int index) {
		return (Shape*)values[index];
	}
};
class devShapeHolder : public AppendableArray {
public:
	__host__ __device__
	devShape* GetShape(int index) {
		return (devShape*)values[index];
	}
};




class LightHolder : public AppendableArray {
public:
	__host__
	void AddLight(Vector3 pos, float intensity) {
		Light* ptr;
		hipMallocManaged(&ptr, sizeof(Light));
		ptr->pos = pos;
		ptr->intensity = intensity;
		AddElement(ptr);
	}
	__host__ __device__
	Light* GetLight(int index) {
		return (Light*)values[index];
	}
};
class devLightHolder : public AppendableArray {
public:
	__host__ __device__
	devLight* GetLight(int index) {
		return (devLight*)values[index];
	}
};