#include "hip/hip_runtime.h"

#include "vectors.cu"
#include <stdio.h>
#include <SDL.h>


#pragma once

struct DistReturn {
	Vector3 col;
	float dist;
};


class Light {
public:
	Vector3 pos;
	float intensity;
	Light(Vector3 newPos) {
		pos = newPos;
	}
};

class Shape {
public:
	char type;
	Transform trans;
	Vector3 col;
	int blend;

	__device__ __host__
		Shape(float x, float y, float z, float rx, float ry, float rz, char typeof, int blended) {
		trans.pos.x = x;
		trans.pos.y = y;
		trans.pos.z = z;

		trans.rot.x = rx;
		trans.rot.y = ry;
		trans.rot.z = rz;

		trans.sca = VONE;

		type = typeof;
		blend = blended;
	}
	__device__ __host__
		Shape(Vector3 newpos, Vector3 newrot, char typeof, int blended) {
		trans.pos = newpos;
		trans.rot = newrot;
		trans.sca = VONE;

		type = typeof;
		blend = blended;
	}
	__device__
		DistReturn DistanceTo(Vector3 currPos) {
		DistReturn result;
		result.col = col;
		result.dist = 1;
		return result;
	};
	__device__
		DistReturn EstimatedDistance(Vector3 currPos) {
		return DistanceTo(currPos);
	}

	__device__ __host__
		Vector3 TransformPoint(Vector3 currpos) {
		return currpos.Sub(trans.pos).ApplyRot(trans.rot.Negative());
	}

	__device__
		virtual Vector3 GetNormal(Vector3 surfacePos) { return VZERO; };
};

class Sphere : public Shape {
public:
	__device__ __host__
		Sphere(float x, float y, float z, float rad, int blended) : Shape(x, y, z, 0, 0, 0, 's', blended) {
		trans.sca = Vector3{ rad, rad, rad };
	}
	__device__ __host__
		Sphere(Vector3 newpos, float rad, int blended) : Shape(newpos, Vector3(0, 0, 0), 's', blended) {
		trans.sca = Vector3{ rad, rad, rad };
	}
	__device__ __host__
		DistReturn DistanceTo(Vector3 currPos) {
		DistReturn result;
		result.dist = trans.pos.Dist(currPos) - trans.sca.x;
		result.col = col;
		return result;
	}
	__device__
		Vector3 GetNormal(Vector3 surfacePos) {
		return surfacePos.Sub(trans.pos).normalised();
	}
};

class Cube : public Shape {
public:
	__device__ __host__
		Cube(float x, float y, float z, float rx, float ry, float rz, float bx, float by, float bz, int blended) : Shape(x, y, z, rx, ry, rz, 'c', blended) {
		trans.sca = Vector3(bx, by, bz);
	}
	__device__ __host__
		Cube(Vector3 newpos, Vector3 newrot, Vector3 bound, int blended) : Shape(newpos, newrot, 'c', blended) {
		trans.sca = bound;
	}
	__device__ __host__
		DistReturn DistanceTo(Vector3 currPos) {
		Vector3 delta = TransformPoint(currPos);
		Vector3 q = delta.abs().Sub(trans.sca);
		DistReturn result;
		delta.x = fmaxf(q.x, 0);
		delta.y = fmaxf(q.y, 0);
		delta.z = fmaxf(q.z, 0);
		result.dist = delta.mag() + fminf(fmaxf(q.x, fmaxf(q.y, q.z)), 0);
		result.col = col;
		return result;
	}
	__device__
		Vector3 GetNormal(Vector3 surfacePos) {
		return surfacePos.Sub(trans.pos).normalised();
	}
	__device__
		DistReturn EstimatedDistance(Vector3 currPos) {
		DistReturn result;
		float rad = trans.sca.mag();
		result.dist = trans.pos.Dist(currPos) - rad;
		return result;
	}
};
class HollowCube : public Shape {
public:
	float thickness;
	__device__ __host__
		HollowCube(float x, float y, float z, float rx, float ry, float rz, float bx, float by, float bz, float e, int blended) : Shape(x, y, z, rx, ry, rz, 'h', blended) {
		trans.sca = Vector3(bx, by, bz);
		thickness = e;
	}
	__device__ __host__
		HollowCube(Vector3 newpos, Vector3 newrot, Vector3 bound, float e, int blended) : Shape(newpos, newrot, 'h', blended) {
		trans.sca = bound;
		thickness = e;
	}
	__device__
		DistReturn DistanceTo(Vector3 currPos) {
		Vector3 delta = TransformPoint(currPos);
		Vector3 p = delta.abs().Sub(trans.sca);
		Vector3 q = p;
		DistReturn result;
		q.x += thickness;
		q.y += thickness;
		q.z += thickness;
		q = q.abs();
		q.x -= thickness;
		q.y -= thickness;
		q.z -= thickness;

		float a = Vector3(p.x, q.y, q.z).Max(Vector3(0, 0, 0)).mag() + fminf(fmaxf(p.x, fmaxf(q.y, q.z)), 0);
		float b = Vector3(q.x, p.y, q.z).Max(Vector3(0, 0, 0)).mag() + fminf(fmaxf(q.x, fmaxf(p.y, q.z)), 0);
		float c = Vector3(q.x, q.y, p.z).Max(Vector3(0, 0, 0)).mag() + fminf(fmaxf(q.x, fmaxf(q.y, p.z)), 0);

		result.dist = fminf(fminf(a, b), c);
		//result.dist = fminf(fminf(a,b), c);
		result.col = col;
		/*
		delta.x = fmaxf(q.x, 0);
		delta.y = fmaxf(q.y, 0);
		delta.z = fmaxf(q.z, 0);
		result.dist = delta.mag() + fminf(fmaxf(q.x, fmaxf(q.y, q.z)), 0);
		result.col = col;*/
		return result;
	}
	__device__
		Vector3 GetNormal(Vector3 surfacePos) {
		return surfacePos.Sub(trans.pos).normalised();
	}
	__device__
		DistReturn EstimatedDistance(Vector3 currPos) {
		DistReturn result;
		float rad = trans.sca.mag();
		result.dist = trans.pos.Dist(currPos) - rad;
		return result;
	}
};

class Plane : public Shape {
public:
	__device__ __host__
		Plane(Vector3 newpos, int blended) : Shape(newpos, VZERO, 'p', blended) { }
	__device__ __host__
		Plane(float height, int blended) : Shape(Vector3(0, height, 0), VZERO, 'p', blended) {}
	__device__
		DistReturn DistanceTo(Vector3 currPos) {
		DistReturn result;
		result.dist = currPos.y - trans.pos.y;
		result.col = col;
		return result;
	}
	__device__
		Vector3 GetNormal(Vector3 surfacePos) {
		return Vector3(0, 1, 0);
	}
};


