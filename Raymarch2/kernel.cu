#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include "engine.cu"

#define MOVE_SPEED 3.0f
#define M_SENS 0.0005f

typedef struct Player {
	Transform trans;
	Vector3 veloc;
} Player;

int main(int argc, char** argv)
{
	//Open the window by initialising renderer object
	Renderer renderer = Renderer(1000, 1000);

	//Initialise camera
	Player player;
	player.trans.pos = Vector3(0, 0, -5);
	player.trans.rot = Vector3(0, 0, 0);

	//Allocate scene memory
	Scene* scene;
	hipMallocManaged(&scene, sizeof(Scene));

	//Initialise primitives and set colours

	Sphere* sphereb = (Sphere*)scene->shapes.CreateSphere(Vector3(0, 3, 0), 0.6f, 0);
	Plane* plane = (Plane*)scene->shapes.CreatePlane(-2.3f, 0);
	Cube* cube = (Cube*)scene->shapes.CreateCube(Vector3(0, -2.3f, 0), Vector3(0, 0, 0), Vector3(3.0f, 0.01f, 100.0f), 0);


	Cube* wall0 = (Cube*)scene->shapes.CreateCube(Vector3(-10, 0, 0), Vector3(0, 0, 0), Vector3(1.0f, 10.f, 10.0f), 0);
	Cube* wall1 = (Cube*)scene->shapes.CreateCube(Vector3(10, 0, 0), Vector3(0, 0, 0), Vector3(1.0f, 10.f, 10.0f), 0);
	Cube* wall2 = (Cube*)scene->shapes.CreateCube(Vector3(0, 0, 10), Vector3(0, 0, 0), Vector3(10.0f, 10.f, 1.0f), 0);
	Cube* wall3 = (Cube*)scene->shapes.CreateCube(Vector3(0, 0, -10), Vector3(0, 0, 0), Vector3(10.0f, 10.f, 1.0f), 0);
	Sphere* spherea = (Sphere*)scene->shapes.CreateSphere(Vector3(-10, 0, 0), 3.0f, 2);

	spherea->col = Vector3(187, 134, 252);
	sphereb->col = Vector3(187, 134, 252);
	plane->col = Vector3(18, 18, 18);
	cube->col = Vector3(50, 50, 50);

	wall0->col = Vector3(255, 0, 0);
	wall1->col = Vector3(255, 0, 0);
	wall2->col = Vector3(255, 0, 0);
	wall3->col = Vector3(255, 0, 0);
	
	//Add light to scene
	scene->lights.AddLight(Vector3(0, 5, 0), 1);

	//Set up timekeeping variables
	int time = SDL_GetTicks();
	float deltaTime;

	//Main loop flag
	bool quit = false;

	//Flag to lock screen to allow for screenshots
	int lockMouse = 0;

	//Event handler
	SDL_Event e;

	//Loop until window is closed
	while (!quit)
	{
		//Iterate over all events
		while (SDL_PollEvent(&e) != 0)
		{
			switch (e.type) {
				//Camera look code
				case (SDL_MOUSEMOTION):
					if (!lockMouse) {
						player.trans.rot.x += e.motion.yrel * M_SENS;
						player.trans.rot.y += e.motion.xrel * M_SENS;
					}
					break;

				//Keydown logic
				case (SDL_KEYDOWN):
					switch (e.key.keysym.sym) {
						case SDLK_w:
							player.veloc.z = MOVE_SPEED;
							break;
						case SDLK_s:
							player.veloc.z = -MOVE_SPEED;
							break;
						case SDLK_a:
							player.veloc.x = -MOVE_SPEED;
							break;
						case SDLK_d:
							player.veloc.x = MOVE_SPEED;
							break;
						case SDLK_e:
							player.veloc.y = MOVE_SPEED;
							break;
						case SDLK_q:
							player.veloc.y = -MOVE_SPEED;
							break;
						case SDLK_SPACE:
							lockMouse = 1 - lockMouse;
							break;
						case SDLK_ESCAPE:
							quit = true;
							break;
						}
					break;

				//Keyup logic
				case (SDL_KEYUP):
					switch (e.key.keysym.sym) {
						case SDLK_w:
							player.veloc.z = 0;
							break;
						case SDLK_s:
							player.veloc.z = 0;
							break;
						case SDLK_a:
							player.veloc.x = 0;
							break;
						case SDLK_d:
							player.veloc.x = 0;
							break;
						case SDLK_e:
							player.veloc.y = 0;
							break;
						case SDLK_q:
							player.veloc.y = 0;
							break;
						}
					break;
			}
			//If quit button pressed
			if (e.type == SDL_QUIT)
			{
				quit = true;
			}
		}

		//Move spheres
		//spherea->trans.pos.x = -1 + -1 * cos(SDL_GetTicks() / 1000.0f) / 3;
		//spherea->trans.pos.y = -0.3f + 1 * cos(SDL_GetTicks() / 1000.0f) / 3;
		sphereb->trans.pos.x = -1 + 1 * cos(SDL_GetTicks() / 1000.0f + 1.58f) / 3;
		sphereb->trans.pos.y = -0.3f + 1 * cos(SDL_GetTicks() / 1000.0f + 1.58f) / 3;
		//spherea->trans.pos = player.trans.pos;
		//Move player
		player.trans.pos = player.trans.pos.Add(player.veloc.ApplyRot(player.trans.rot).Mul(deltaTime));

		//Position light above player
		scene->lights.GetLight(0)->pos = player.trans.pos.Add(Vector3(0, 0, 0));

		//Draw frame
		renderer.Draw(player.trans, *scene);

		//Calculate time between frames
		deltaTime = (SDL_GetTicks() - time) / 1000.0f;

		//Debug text
		//printf("Time for frame: %ums\n", SDL_GetTicks() - time);
		printf("Distance to sphere: %f\n", subDist(wall0->DistanceTo(player.trans.pos), spherea->DistanceTo(player.trans.pos)).dist);
		//Update time since program start
		time = SDL_GetTicks();
	}

	return 0;
}